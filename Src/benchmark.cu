/****************************************************************************************/
/* Benchmark script for GEMM                                                            */
/* -------------------------                                                            */
/* Author: Mathias Otnes                                                                */
/* year:   2024                                                                         */
/****************************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "gemm.h"

int main()
{
    // CUDA event objects
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start recording time
    hipEventRecord(start);
	printf("\nHello world!\n");
    
    // Stop recording time
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0.0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Time elapsed to print hello world: %f ms\n", milliseconds);
	
	return 0;
} 
