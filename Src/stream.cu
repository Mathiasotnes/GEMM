#include "hip/hip_runtime.h"
/********************************************************************/
/* stream.cu                                                        */
/* ---------------------------------------------------------------- */
/* Naive implementation of GEMM using streams for memory allocation */
/* ---------------------------------------------------------------- */
/* Author: Mathias Otnes                                            */
/* year:   2024                                                     */
/********************************************************************/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> 
#include <stdio.h>
#include "gemm.h"

#define STREAMS 4

__global__ void gemm_stream_kernel( float* d_a, float* d_b, float* d_c, int N )
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N)
    {
        float sum = 0.0f;
        for (int i = 0; i < N; i++)
        {
            sum += d_a[row * N + i] * d_b[i * N + col];
        }
        d_c[row * N + col] = sum;
    }
}

/**
 * @brief Uses streams for memory allocation and kernel execution
 * 
 * @param A Matrix A
 * @param B Matrix B
 * @param C Matrix C
 * @param N Size of matrices
 */
void gemm_stream( float* A, float* B, float* C, int N ) 
{

    float *d_a[STREAMS], *d_b[STREAMS], *d_c[STREAMS];
    int streamSize = N / STREAMS;
	int streamBytes = streamSize * sizeof( int );

    if ( VERBOSE ) {
        printf("Launching streamed kernel with %d streams\n", STREAMS);
    }

    hipStream_t stream[STREAMS];
    for ( int i = 0; i < STREAMS; i++ )
    {
        hipStreamCreate(&stream[i]);
    }

	// Allocate memory on device
	for ( int i = 0; i < STREAMS; i++ )
    {
        checkCudaErrors( hipMalloc(&d_a[i], streamBytes) );
        checkCudaErrors( hipMalloc(&d_b[i], streamBytes) );
        checkCudaErrors( hipMalloc(&d_c[i], streamBytes) );
    }

    // Enable DMA transfer operation by allocating pinned host memory
    checkCudaErrors( hipHostMalloc((void **)&A, N * N * sizeof(float)) );
    checkCudaErrors( hipHostMalloc((void **)&B, N * N * sizeof(float)) );
    checkCudaErrors( hipHostMalloc((void **)&C, N * N * sizeof(float)) );

	// Copy data to device and start streams asynchronously
    for ( int i = 0; i < STREAMS; i++ )
    {
        checkCudaErrors( hipMemcpyAsync(d_a[i], A + i * streamSize, streamBytes, hipMemcpyHostToDevice, stream[i]) );
        checkCudaErrors( hipMemcpyAsync(d_b[i], B + i * streamSize, streamBytes, hipMemcpyHostToDevice, stream[i]) );

        dim3 blockSize(16, 16);
        dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

        if ( VERBOSE ) {
            printf("Launching streamed kernel %d with grid size %d, %d and block size %d, %d\n", i, gridSize.x, gridSize.y, blockSize.x, blockSize.y);
        }

        gemm_stream_kernel<<<gridSize, blockSize, 0, stream[i]>>>(d_a[i], d_b[i], d_c[i], N);

        checkCudaErrors( hipMemcpyAsync(C + i * streamSize, d_c[i], streamBytes, hipMemcpyDeviceToHost, stream[i]) );

    }

    // Wait for the streams to finish
    for ( int i = 0; i < STREAMS; i++ )
    {
        checkCudaErrors( hipStreamSynchronize(stream[i]) );
    }

    checkCudaErrors( hipDeviceSynchronize() );

    for ( int i = 0; i < STREAMS; i++ )
    {
        checkCudaErrors( hipFree(d_a[i]) );
        checkCudaErrors( hipFree(d_b[i]) );
        checkCudaErrors( hipFree(d_c[i]) );
        checkCudaErrors( hipStreamDestroy(stream[i]) );
    }

}
