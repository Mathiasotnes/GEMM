/********************************************************************************************************/
/* cublas.cu                                                                                            */
/* ----------------------                                                                               */
/* Wrapper for CBLAS GEMM                                                                               */
/* Inspired by NVIDIA sample:                                                                           */
/* https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuBLAS/Level-3/gemm/cublas_gemm_example.cu  */
/* ----------------------                                                                               */
/* Author: Mathias Otnes                                                                                */
/* year:   2024                                                                                         */
/********************************************************************************************************/

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>



#include <stdio.h>

void gemm_cublas_kernel( float* A_d, float* B_d, float* C_d, int N )
{
    hipblasHandle_t handle;
    hipStream_t stream;

    // Create cuBLAS handle and stream
    hipblasCreate(&handle);
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipblasSetStream(handle, stream);

    // Configure SGEMM
    float alpha     = 1.0f;
    float beta      = 0.0f;
    int   lda       = N; 
    int   ldb       = N; 
    int   ldc       = N;

    // To compute C = A * B in row-major, we call cublasSgemm with B and A swapped
    hipblasStatus_t status = hipblasSgemm(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,  // No transpose for both
        N, N, N,                   // M, N, K
        &alpha,
        B_d, ldb,                  // Note that B_d comes first (swap order)
        A_d, lda,                  // A_d comes second
        &beta,
        C_d, ldc                   // C matrix
    );

    if ( status != HIPBLAS_STATUS_SUCCESS ) {
        printf("cuBLAS SGEMM failed\n");
    }

    hipStreamSynchronize(stream);

    hipblasDestroy(handle);
    hipStreamDestroy(stream);
}

void gemm_cublas( float* A, float* B, float* C, int N ) 
{

    // Allocate memory on device
    float *A_d, *B_d, *C_d;
    hipMalloc(&A_d, N * N * sizeof(float));
    hipMalloc(&B_d, N * N * sizeof(float));
    hipMalloc(&C_d, N * N * sizeof(float));

    // Copy data to device
    hipMemcpy(A_d, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Run kernel
    gemm_cublas_kernel(A_d, B_d, C_d, N);

    // Copy data back to host
    hipMemcpy(C, C_d, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Free memory on device
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

}
