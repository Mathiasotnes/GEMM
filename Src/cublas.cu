/********************************************************************************************************/
/* cublas.cu                                                                                            */
/* ----------------------                                                                               */
/* Wrapper for CBLAS GEMM                                                                               */
/* Inspired by NVIDIA sample:                                                                           */
/* https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuBLAS/Level-3/gemm/cublas_gemm_example.cu  */
/* ----------------------                                                                               */
/* Author: Mathias Otnes                                                                                */
/* year:   2024                                                                                         */
/********************************************************************************************************/

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

void gemm_cublas(float* A_d, float* B_d, float* C_d, int N)
{
    hipblasHandle_t handle;
    hipStream_t stream;

    // Step 1: Create cuBLAS handle and bind a stream
    hipblasCreate(&handle);
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipblasSetStream(handle, stream);

    // Step 2: Set up constants and leading dimensions
    float alpha = 1.0f;
    float beta = 0.0f;
    int lda = N;
    int ldb = N;
    int ldc = N;

    // Step 3: Call cuBLAS SGEMM
    // cuBLAS is column-major by default. Since you're working with row-major matrices,
    // transpose both matrices.
    hipblasStatus_t status = hipblasSgemm(
        handle,
        HIPBLAS_OP_T, HIPBLAS_OP_T,  // Transpose both A and B for row-major compatibility
        N, N, N,                   // M, N, K
        &alpha,                    // Scalar for multiplication
        B_d, ldb,                  // B device pointer, leading dimension ldb
        A_d, lda,                  // A device pointer, leading dimension lda
        &beta,                     // Scalar for accumulation
        C_d, ldc                   // C device pointer, leading dimension ldc
    );

    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS SGEMM failed\n");
    }

    // Step 4: Synchronize
    hipStreamSynchronize(stream);

    // Step 5: Clean up
    hipblasDestroy(handle);
    hipStreamDestroy(stream);
}
