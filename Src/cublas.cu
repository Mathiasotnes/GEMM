/********************************************************************************************************/
/* cublas.cu                                                                                            */
/* ----------------------                                                                               */
/* Wrapper for CBLAS GEMM                                                                               */
/* Inspired by NVIDIA sample:                                                                           */
/* https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuBLAS/Level-3/gemm/cublas_gemm_example.cu  */
/* ----------------------                                                                               */
/* Author: Mathias Otnes                                                                                */
/* year:   2024                                                                                         */
/********************************************************************************************************/

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>



#include <stdio.h>

void gemm_cublas(float* A_d, float* B_d, float* C_d, int N)
{
    hipblasHandle_t handle;
    hipStream_t stream;

    // Step 1: Create cuBLAS handle and bind a stream
    hipblasCreate(&handle);
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipblasSetStream(handle, stream);

    float alpha = 1.0f;
    float beta = 0.0f;
    int lda = N, ldb = N, ldc = N;

    // Step 2: Call cuBLAS SGEMM with swapped order
    // To compute C = A * B in row-major, we call cublasSgemm with B and A swapped
    hipblasStatus_t status = hipblasSgemm(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,  // No transpose for both
        N, N, N,                   // M, N, K
        &alpha,
        B_d, ldb,                  // Note that B_d comes first (swap order)
        A_d, lda,                  // A_d comes second
        &beta,
        C_d, ldc                   // C matrix
    );

    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS SGEMM failed\n");
    }

    // Step 3: Synchronize to ensure computation is complete
    hipStreamSynchronize(stream);

    // Step 4: Clean up
    hipblasDestroy(handle);
    hipStreamDestroy(stream);
}

