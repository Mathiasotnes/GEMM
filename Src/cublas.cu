/********************************************************************************************************/
/* cublas.cu                                                                                            */
/* ---------------------------------------------------------------------------------------------------- */
/* Wrapper for CBLAS GEMM                                                                               */
/* Inspired by NVIDIA sample:                                                                           */
/* https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuBLAS/Level-3/gemm/cublas_gemm_example.cu  */
/* ---------------------------------------------------------------------------------------------------- */
/* Author: Mathias Otnes                                                                                */
/* year:   2024                                                                                         */
/********************************************************************************************************/

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

void gemm_cublas_kernel( float* A_d, float* B_d, float* C_d, int N )
{
    hipblasHandle_t  handle;
    hipStream_t    stream;

    hipblasCreate(&handle);
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipblasSetStream(handle, stream);

    // Configure SGEMM to match our problem
    float alpha     = 1.0f;
    float beta      = 0.0f;
    int   lda       = N; 
    int   ldb       = N; 
    int   ldc       = N;

    // To compute C = A * B in row-major, we call cublasSgemm with B and A swapped, because
    // cuBLAS assumes column-major order.
    hipblasStatus_t status = hipblasSgemm(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        N, N, N,
        &alpha,
        B_d, ldb, // Note that B_d comes before A_d
        A_d, lda,
        &beta,
        C_d, ldc
    );

    if ( status != HIPBLAS_STATUS_SUCCESS ) {
        printf("cuBLAS SGEMM failed\n");
    }

    hipStreamSynchronize(stream);

    hipblasDestroy(handle);
    hipStreamDestroy(stream);
}

void gemm_cublas( float* A, float* B, float* C, int N ) 
{

    // Memory allocation
    float *A_d, *B_d, *C_d;
    hipMalloc(&A_d, N * N * sizeof(float));
    hipMalloc(&B_d, N * N * sizeof(float));
    hipMalloc(&C_d, N * N * sizeof(float));

    // Host -> Device
    hipMemcpy(A_d, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Run kernel
    gemm_cublas_kernel(A_d, B_d, C_d, N);

    // Device -> Host
    hipMemcpy(C, C_d, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Memory deallocation
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}
