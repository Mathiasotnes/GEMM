/********************************************************************/
/* naive.cu                                                         */
/* ----------------------------                                     */
/* Naive implementation of GEMM                                     */
/* ----------------------------                                     */
/* Author: Mathias Otnes                                            */
/* year:   2024                                                     */
/********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void gemm_naive_kernel( float* A_d, float* B_d, float* C_d, int N )
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N)
    {
        float sum = 0.0f;
        for (int i = 0; i < N; i++)
        {
            sum += A_d[row * N + i] * B_d[i * N + col];
        }
        C_d[row * N + col] = sum;
    }
}

void gemm_naive( float* A, float* B, float* C, int N ) 
{

	// Allocate memory on device
	float *A_d, *B_d, *C_d;
	hipMalloc(&A_d, N * N * sizeof(float));
	hipMalloc(&B_d, N * N * sizeof(float));
	hipMalloc(&C_d, N * N * sizeof(float));

	// Copy data to device
	hipMemcpy(A_d, A, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, N * N * sizeof(float), hipMemcpyHostToDevice);

	// Kernel configuration
    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

	// Run kernel
    gemm_naive_kernel<<<gridSize, blockSize>>>(A_d, B_d, C_d, N);
    hipDeviceSynchronize();

	// Copy data back to host
	hipMemcpy(C, C_d, N * N * sizeof(float), hipMemcpyDeviceToHost);

	// Free memory on device
	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);

}
