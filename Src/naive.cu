#include "hip/hip_runtime.h"
/****************************************************************************************/
/* naive.cu                                                                             */
/* ------------------------------------------------------------------------------------ */
/* Naive implementation of GEMM                                                 		*/
/* ------------------------------------------------------------------------------------ */
/* Author: Mathias Otnes                                                                */
/* year:   2024                                                                         */
/*                                                                                      */
/* Inspiration:                                                                         */
/* - https://leimao.github.io/article/CUDA-Matrix-Multiplication-Optimization/          */
/*                                                                                      */
/****************************************************************************************/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> 
#include <stdio.h>
#include "gemm.h"

__global__ void gemm_naive_kernel( float* A_d, float* B_d, float* C_d, int N )
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if ( row < N && col < N ) {
        float sum = 0.0f;
        for ( int i = 0; i < N; i++ ) {
            sum += A_d[row * N + i] * B_d[i * N + col];
        }
        C_d[row * N + col] = sum;
    }
}

void gemm_naive( float* A, float* B, float* C, int N ) 
{

	// Memory allocation
	float *A_d, *B_d, *C_d;

	checkCudaErrors( hipMalloc(&A_d, N * N * sizeof(float)) );
	checkCudaErrors( hipMalloc(&B_d, N * N * sizeof(float)) );
	checkCudaErrors( hipMalloc(&C_d, N * N * sizeof(float)) );

	// Host -> Device
	checkCudaErrors( hipMemcpy(A_d, A, N * N * sizeof(float), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(B_d, B, N * N * sizeof(float), hipMemcpyHostToDevice) );

	// Launch kernel
    dim3 blockSize(TILE_SIZE, TILE_SIZE);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

	if ( VERBOSE ) {
		printf("Launching naive kernel with grid size %d, %d and block size %d, %d\n", gridSize.x, gridSize.y, blockSize.x, blockSize.y);
	}

    gemm_naive_kernel<<<gridSize, blockSize>>>(A_d, B_d, C_d, N);

	hipError_t err = hipGetLastError();
	if ( err != hipSuccess ) {
		printf("Kernel launch error: %s\n", hipGetErrorString(err));
	}

    checkCudaErrors( hipDeviceSynchronize() );

	// Device -> Host
	checkCudaErrors( hipMemcpy(C, C_d, N * N * sizeof(float), hipMemcpyDeviceToHost) );

	// Memory deallocation
	checkCudaErrors( hipFree(A_d) );
	checkCudaErrors( hipFree(B_d) );
	checkCudaErrors( hipFree(C_d) );

}
