#include "hip/hip_runtime.h"
/****************************************************************************************/
/* stream_shmem.cu                                                                      */
/* --------------------------------                                                     */
/* Optimized implementation of GEMM                                                     */
/* --------------------------------                                                     */
/* Author: Mathias Otnes                                                                */
/* year:   2024                                                                         */
/*                                                                                      */
/* Inspirations:                                                                        */
/* - https://leimao.github.io/article/CUDA-Matrix-Multiplication-Optimization/          */
/*                                                                                      */
/****************************************************************************************/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> 
#include <stdio.h>
#include "gemm.h"

#define TILE_SIZE 16

__global__ void gemm_stream_shmem_kernel( float* A_tile, float* B_tile, float* C, int N, int C_row_offset, int C_col_offset )
{
    __shared__ float tile_A[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_B[TILE_SIZE][TILE_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Load tiles into shared memory
    tile_A[ty][tx] = A_tile[ty * TILE_SIZE + tx];
    tile_B[ty][tx] = B_tile[ty * TILE_SIZE + tx];
    __syncthreads();

    float val = 0.0f;

    // Compute the dot product for the tile
    for (int k = 0; k < TILE_SIZE; k++)
    {
        val += tile_A[ty][k] * tile_B[k][tx];
    }

    // Calculate global row and column indices
    int row = C_row_offset + ty;
    int col = C_col_offset + tx;

    // Write the result to the global matrix C
    if (row < N && col < N) {
        C[row * N + col] += val;
    }
}

void gemm_stream_shmem( float* A, float* B, float* C, int N )
{

    size_t matrix_size = N * N * sizeof(float);

    // Calculate the number of tiles
    int num_tiles = (N + TILE_SIZE - 1) / TILE_SIZE;
    int num_streams = num_tiles;

    // Allocate device memory for C
    float *C_d;
    checkCudaErrors(hipMalloc((void**)&C_d, matrix_size));
    checkCudaErrors(hipMemcpy(C_d, C, matrix_size, hipMemcpyHostToDevice));

    // Allocate arrays for tiles and streams
    float **tile_A = new float*[num_streams];
    float **tile_B = new float*[num_streams];
    hipStream_t *streams = new hipStream_t[num_streams];

    // Allocate device memory for tiles and create streams
    for (int i = 0; i < num_streams; i++)
    {
        checkCudaErrors(hipMalloc((void**)&tile_A[i], TILE_SIZE * TILE_SIZE * sizeof(float)));
        checkCudaErrors(hipMalloc((void**)&tile_B[i], TILE_SIZE * TILE_SIZE * sizeof(float)));
        checkCudaErrors(hipStreamCreate(&streams[i]));
    }

    int stream_idx = 0;
    for (int row = 0; row < num_tiles; row++)
    {
        for (int col = 0; col < num_tiles; col++)
        {
            hipStream_t stream = streams[stream_idx];

            // Calculate tile offsets
            int C_row_offset = row * TILE_SIZE;
            int C_col_offset = col * TILE_SIZE;
            int A_row_offset = C_row_offset;
            int A_col_offset = 0;
            int B_row_offset = 0;
            int B_col_offset = C_col_offset;

            // Copy tiles to device
            // Copy tile from A
            for (int i = 0; i < TILE_SIZE; i++)
            {
                int A_row = A_row_offset + i;
                if (A_row < N)
                {
                    checkCudaErrors(hipMemcpyAsync(
                        tile_A[stream_idx] + i * TILE_SIZE,
                        A + A_row * N + A_col_offset,
                        TILE_SIZE * sizeof(float),
                        hipMemcpyHostToDevice, stream));
                }
                else
                {
                    checkCudaErrors(hipMemsetAsync(tile_A[stream_idx] + i * TILE_SIZE, 0, TILE_SIZE * sizeof(float), stream));
                }
            }

            // Copy tile from B
            for (int i = 0; i < TILE_SIZE; i++)
            {
                int B_row = B_row_offset + i;
                if (B_row < N)
                {
                    checkCudaErrors(hipMemcpyAsync(
                        tile_B[stream_idx] + i * TILE_SIZE,
                        B + B_row * N + B_col_offset,
                        TILE_SIZE * sizeof(float),
                        hipMemcpyHostToDevice, stream));
                }
                else
                {
                    checkCudaErrors(hipMemsetAsync(tile_B[stream_idx] + i * TILE_SIZE, 0, TILE_SIZE * sizeof(float), stream));
                }
            }

            // Kernel launch parameters
            dim3 blockSize(TILE_SIZE, TILE_SIZE);
            dim3 gridSize(1, 1);

            // Launch kernel
            gemm_stream_shmem_kernel<<<gridSize, blockSize, 0, stream>>>(tile_A[stream_idx], tile_B[stream_idx], C_d, N, C_row_offset, C_col_offset);

            // Update stream index
            stream_idx = (stream_idx + 1) % num_streams;
        }
    }

    // Synchronize streams
    for (int i = 0; i < num_streams; ++i)
    {
        checkCudaErrors(hipStreamSynchronize(streams[i]));
    }

    // Copy result back to host
    checkCudaErrors(hipMemcpy(C, C_d, matrix_size, hipMemcpyDeviceToHost));

    // Free device memory and destroy streams
    for (int i = 0; i < num_streams; i++)
    {
        checkCudaErrors(hipFree(tile_A[i]));
        checkCudaErrors(hipFree(tile_B[i]));
        checkCudaErrors(hipStreamDestroy(streams[i]));
    }
    delete[] tile_A;
    delete[] tile_B;
    delete[] streams;

    checkCudaErrors(hipFree(C_d));
}
